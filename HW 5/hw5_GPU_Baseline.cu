#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define ALIVE 1
#define DEAD 0

__global__ void updateBoard_GlobalMemory(int* grid, int* nextGrid, int* flag, int N) {
    int global_x = blockIdx.x * blockDim.x + threadIdx.x, global_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (global_x && global_x < N - 1 && global_y && global_y < N - 1) {
        int neighbors = grid[(global_y - 1) * N + (global_x - 1)] + grid[(global_y - 1) * N + global_x] + grid[(global_y - 1) * N + (global_x + 1)] + grid[global_y * N + (global_x - 1)] + grid[global_y * N + (global_x + 1)] + grid[(global_y + 1) * N + (global_x - 1)] + grid[(global_y + 1) * N + global_x] + grid[(global_y + 1) * N + (global_x + 1)];

        if (grid[global_y * N + global_x]) {
            if (neighbors < 2 || neighbors > 3) {
                nextGrid[global_y * N + global_x] = DEAD;
                atomicAdd(flag, 1);
            } 
            else {
                nextGrid[global_y * N + global_x] = ALIVE;
            }
        } 
        else {
            if (neighbors == 3) {
                nextGrid[global_y * N + global_x] = ALIVE;
                atomicAdd(flag, 1);
            } 
            else {
                nextGrid[global_y * N + global_x] = DEAD;
            }
        }
    }
}

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void printGrid(int* grid, int N) {
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            printf("%d ", grid[i * N + j]);
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {
    int N = atoi(argv[1]) + 2, maxGenerations = atoi(argv[2]);
    const int blockSize = 32;

    char outputDirectory[150];
    sprintf(outputDirectory, "%s/output.%s.%s.GPU.Baseline.txt", argv[3], argv[1], argv[2]);

    int* hostGrid = (int*)malloc(N * N * sizeof(int));
    int* hostNextGrid = (int*)malloc(N * N * sizeof(int));

    for (int i = 0; i < N; i++) {
        srand(54321 | i);
        for (int j = 0; j < N; j++) {
            if (i == 0 || i == N - 1 || j == 0 || j == N - 1) {
                hostGrid[i * N + j] = DEAD;
            } 
            else {
                if (drand48() < 0.5) 
                    hostGrid[i * N + j] = ALIVE;
                else 
                    hostGrid[i * N + j] = DEAD;
            }
        }
    }

    #ifdef DEBUG
        printf("Initial Grid:\n");
        printGrid(hostGrid, N);
    #endif
    

    hipEvent_t startEvent, stopEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    float ms;

    int* deviceCurrentGrid;
    int* deviceNextGrid;
    int* deviceFlag;
    checkCuda(hipMalloc((void**)&deviceCurrentGrid, N * N * sizeof(int)));
    checkCuda(hipMalloc((void**)&deviceNextGrid, N * N * sizeof(int)));
    checkCuda(hipMalloc((void**)&deviceFlag, sizeof(int)));

    checkCuda(hipMemcpy(deviceCurrentGrid, hostGrid, N * N * sizeof(int), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 blocksPerGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    checkCuda(hipEventRecord(startEvent));

    for (int gen = 0; gen < maxGenerations; gen++) {
        checkCuda(hipMemset(deviceFlag, 0, sizeof(int)));
        
        updateBoard_GlobalMemory<<<blocksPerGrid, threadsPerBlock>>>(deviceCurrentGrid, deviceNextGrid, deviceFlag, N);

        int flag;
        checkCuda(hipMemcpy(&flag, deviceFlag, sizeof(int), hipMemcpyDeviceToHost));
        if (!flag) {
            printf("Converged at generation %d\n", gen + 1);
            break;
        }

        int* temp = deviceCurrentGrid;
        deviceCurrentGrid = deviceNextGrid;
        deviceNextGrid = temp;
    }

    checkCuda(hipMemcpy(hostGrid, deviceCurrentGrid, N * N * sizeof(int), hipMemcpyDeviceToHost));
    
    checkCuda(hipEventRecord(stopEvent));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));

    #ifdef DEBUG
        printf("Final Grid:\n");
        printGrid(hostGrid, N);
    #endif

    FILE* outputFile = fopen(outputDirectory, "w");
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            fprintf(outputFile, "%d ", hostGrid[i * N + j]);
        }
        fprintf(outputFile, "\n");
    }
    fclose(outputFile);

    printf("GPU Baseline Algorithm of the Game of Life: Matrix of size %d x %d with %d generations computed in %f sec\n\n", N - 2, N - 2, maxGenerations, ms / 1000);

    checkCuda(hipFree(deviceCurrentGrid));
    checkCuda(hipFree(deviceNextGrid));
    checkCuda(hipFree(deviceFlag));

    free(hostGrid);
    free(hostNextGrid);

    return 0;
}