#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define ALIVE 1
#define DEAD 0

__global__ void updateBoard(int* grid, int* nextGrid, int* flag, int N) {
    extern __shared__ int sharedGrid[];

    // Global indices
    int global_x = blockIdx.x * blockDim.x + threadIdx.x, global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Shared memory dims
    int shared_size = blockDim.x + 2;

    // Load current cell
    if (global_x < N && global_y < N) {
        sharedGrid[(threadIdx.y + 1) * shared_size + (threadIdx.x + 1)] = grid[global_y * N + global_x];

        // Load ghost cells, top row
        if (threadIdx.y == DEAD) {
            if (global_y)
                sharedGrid[threadIdx.y * shared_size + (threadIdx.x + 1)] = grid[(global_y - 1) * N + global_x];

            // Load ghost cells, top left corner
            if (threadIdx.x == DEAD && global_x && global_y) 
                sharedGrid[threadIdx.y * shared_size + threadIdx.x] = grid[(global_y - 1) * N + (global_x - 1)];

            // Load ghost cells, top right corner
            if (threadIdx.x == blockDim.x - 1 && global_y && global_x < N - 1)
                sharedGrid[threadIdx.y * shared_size + (threadIdx.x + 2)] = grid[(global_y - 1) * N + (global_x + 1)];
        }

        // Load ghost cells, bottom row
        if (threadIdx.y == blockDim.y - 1) {
            if (global_y < N - 1)
                sharedGrid[(threadIdx.y + 2) * shared_size + (threadIdx.x + 1)] = grid[(global_y + 1) * N + global_x];

            // Load ghost cells, bottom left corner
            if (threadIdx.x == DEAD && global_x && global_y < N - 1) 
                sharedGrid[(threadIdx.y + 2) * shared_size + threadIdx.x] = grid[(global_y + 1) * N + (global_x - 1)];

            // Load ghost cells, bottom right corner
            if (threadIdx.x == blockDim.x - 1 && global_x < N - 1 && global_y < N - 1) 
                sharedGrid[(threadIdx.y + 2) * shared_size + (threadIdx.x + 2)] = grid[(global_y + 1) * N + (global_x + 1)];
        }

        // Load ghost cells, left column
        if (threadIdx.x == DEAD && global_x)
            sharedGrid[(threadIdx.y + 1) * shared_size + threadIdx.x] = grid[global_y * N + (global_x - 1)];

        // Load ghost cells, right column
        if (threadIdx.x == blockDim.x - 1 && global_x < N - 1)
            sharedGrid[(threadIdx.y + 1) * shared_size + (threadIdx.x + 2)] = grid[global_y * N + (global_x + 1)];
    }

    __syncthreads();

    // Update current cell
    if (global_x && global_x < N - 1 && global_y && global_y < N - 1) {
        int neighbors = sharedGrid[threadIdx.y * shared_size + threadIdx.x] + sharedGrid[threadIdx.y * shared_size + (threadIdx.x + 1)] + sharedGrid[threadIdx.y * shared_size + (threadIdx.x + 2)] + sharedGrid[(threadIdx.y + 1) * shared_size + threadIdx.x] + sharedGrid[(threadIdx.y + 1) * shared_size + (threadIdx.x + 2)] + sharedGrid[(threadIdx.y + 2) * shared_size + threadIdx.x] + sharedGrid[(threadIdx.y + 2) * shared_size + (threadIdx.x + 1)] + sharedGrid[(threadIdx.y + 2) * shared_size + (threadIdx.x + 2)];

        if (sharedGrid[(threadIdx.y + 1) * shared_size + (threadIdx.x + 1)]) {
            if (neighbors < 2 || neighbors > 3) {
                nextGrid[global_y * N + global_x] = DEAD;
                atomicAdd(flag, 1);
            } 
            else {
                nextGrid[global_y * N + global_x] = ALIVE;
            }
        } 
        else {
            if (neighbors == 3) {
                nextGrid[global_y * N + global_x] = ALIVE;
                atomicAdd(flag, 1);
            } 
            else {
                nextGrid[global_y * N + global_x] = DEAD;
            }
        }
    }
}

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void printGrid(int* grid, int N) {
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            printf("%d ", grid[i * N + j]);
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {
    int N = atoi(argv[1]) + 2, maxGenerations = atoi(argv[2]);
    const int blockSize = 32;

    char outputDirectory[150];
    sprintf(outputDirectory, "%s/output.%s.%s.GPU.Shared.txt", argv[3], argv[1], argv[2]);

    int* hostGrid = (int*)malloc(N * N * sizeof(int));
    int* hostNextGrid = (int*)malloc(N * N * sizeof(int));

    for (int i = 0; i < N; i++) {
        srand(54321 | i);
        for (int j = 0; j < N; j++) {
            if (i == 0 || i == N - 1 || j == 0 || j == N - 1) {
                hostGrid[i * N + j] = DEAD;
            } 
            else {
                if (drand48() < 0.5) 
                    hostGrid[i * N + j] = ALIVE;
                else 
                    hostGrid[i * N + j] = DEAD;
            }
        }
    }

    #ifdef DEBUG
        printf("Initial Grid:\n");
        printGrid(hostGrid, N);
    #endif
    

    hipEvent_t startEvent, stopEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    float ms;

    int* deviceCurrentGrid;
    int* deviceNextGrid;
    int* deviceFlag;
    checkCuda(hipMalloc((void**)&deviceCurrentGrid, N * N * sizeof(int)));
    checkCuda(hipMalloc((void**)&deviceNextGrid, N * N * sizeof(int)));
    checkCuda(hipMalloc((void**)&deviceFlag, sizeof(int)));

    checkCuda(hipMemcpy(deviceCurrentGrid, hostGrid, N * N * sizeof(int), hipMemcpyHostToDevice));

    // N = matrix size + 2 for ghost cells
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 blocksPerGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    checkCuda(hipEventRecord(startEvent));

    for (int gen = 0; gen < maxGenerations; gen++) {
        checkCuda(hipMemset(deviceFlag, 0, sizeof(int)));
        
        updateBoard<<<blocksPerGrid, threadsPerBlock, (blockSize + 2) * (blockSize + 2) * sizeof(int)>>>(deviceCurrentGrid, deviceNextGrid, deviceFlag, N);

        int flag;
        checkCuda(hipMemcpy(&flag, deviceFlag, sizeof(int), hipMemcpyDeviceToHost));
        if (!flag) {
            printf("Converged at generation %d\n", gen + 1);
            break;
        }

        int* temp = deviceCurrentGrid;
        deviceCurrentGrid = deviceNextGrid;
        deviceNextGrid = temp;
    }

    checkCuda(hipMemcpy(hostGrid, deviceCurrentGrid, N * N * sizeof(int), hipMemcpyDeviceToHost));
    
    checkCuda(hipEventRecord(stopEvent));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));

    #ifdef DEBUG
        printf("Final Grid:\n");
        printGrid(hostGrid, N);
    #endif

    FILE* outputFile = fopen(outputDirectory, "w");
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            fprintf(outputFile, "%d ", hostGrid[i * N + j]);
        }
        fprintf(outputFile, "\n");
    }
    fclose(outputFile);

    printf("GPU Shared Memory Optimization of the Game of Life: Matrix of size %d x %d with %d generations computed in %f sec\n\n", N - 2, N - 2, maxGenerations, ms / 1000);

    checkCuda(hipFree(deviceCurrentGrid));
    checkCuda(hipFree(deviceNextGrid));
    checkCuda(hipFree(deviceFlag));

    free(hostGrid);
    free(hostNextGrid);

    return 0;
}